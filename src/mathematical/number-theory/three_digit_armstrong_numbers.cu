#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <iostream>

// Define global constants in host memory
constexpr unsigned NUM_DIGITS = 10;
constexpr unsigned MIN_NUMBER = 99;
constexpr unsigned MAX_NUMBER = 1000;
constexpr unsigned BLOCK_DIM = 1 << 5;
constexpr unsigned ARRAY_DIM = 1 << 10;
constexpr unsigned NUM_STREAMS = 1 << 1;
constexpr unsigned ARRAY_BYTES = ARRAY_DIM * sizeof(bool);
constexpr unsigned NUM_ELEMENTS_PER_STREAM = ARRAY_DIM / NUM_STREAMS;
constexpr unsigned CUBED_DIGITS_BYTES = NUM_DIGITS * sizeof(unsigned);
constexpr unsigned CUBED_DIGITS[NUM_DIGITS] = {0, 1, 8, 27, 64, 125, 216, 343, 512, 729};

// Define global array in device constant memory
__constant__ unsigned CUBED_DIGITS_DEVICE_CONSTANT[NUM_DIGITS] = {0, 1, 8, 27, 64, 125, 216, 343, 512, 729};

// Declare global array reference to device texture memory
texture<unsigned, 1, hipReadModeElementType> CUBED_DIGITS_DEVICE_TEXTURE;

// Define three-digit Armstrong numbers calculation kernel
__global__ void ThreeDigitArmstrongNumbersKernel(bool *array, unsigned streamIdx) {
    unsigned number = NUM_ELEMENTS_PER_STREAM * streamIdx + blockIdx.x * blockDim.x + threadIdx.x;
    array[number] = false;
    if (number > MIN_NUMBER && number < MAX_NUMBER) {
        unsigned sumOfCubedDigits = 0;
        for (unsigned digit, digits = number; digits > 0; digits /= 10) {
            digit = digits % 10;
            // Read pre-computed data from constant device memory
            // sumOfCubedDigits += CUBED_DIGITS_DEVICE_CONSTANT[digit];
            // Read pre-computed data from texture device memory
            sumOfCubedDigits += tex1Dfetch(CUBED_DIGITS_DEVICE_TEXTURE, digit);
        }
        array[number] = (sumOfCubedDigits == number);
    }
}

int main() {
    std::cout << "HOST EXECUTION\n";

    // Declare host clock variables
    float elapsedTimeHost;
    clock_t startTimeHost, stopTimeHost;

    // Start host clock
    startTimeHost = clock();

    // Compute three-digit Armstrong numbers on host as on device
    std::cout << "Three-digit Armstrong numbers computed on host: ";
    unsigned sumOfCubedDigits;
    for (unsigned number = 0; number < ARRAY_DIM; ++number) {
        if (number > MIN_NUMBER && number < MAX_NUMBER) {
            sumOfCubedDigits = 0;
            for (unsigned digit, digits = number; digits > 0; digits /= 10) {
                digit = digits % 10;
                sumOfCubedDigits += CUBED_DIGITS[digit];
            }
            if (sumOfCubedDigits == number) {
                std::cout << number << ' ';
            }
        }
    }
    std::cout << '\n';

    // Stop host clock
    stopTimeHost = clock();
    elapsedTimeHost = stopTimeHost - startTimeHost;
    std::cout << "Elapsed Time on Host: " << elapsedTimeHost << " ms\n\n";

    std::cout << "DEVICE EXECUTION\n";

    // Declare array for output data on host
    bool hostArray[ARRAY_DIM];

    // Declare pointer to output data on device
    bool *deviceArray = nullptr;

    // Declare pointer to pre-computed data on device
    unsigned *deviceCubedDigits = nullptr;

    // Allocate device memory for output and pre-computed data
    hipMalloc(&deviceArray, ARRAY_BYTES);
    hipMalloc(&deviceCubedDigits, CUBED_DIGITS_BYTES);

    // Copy pre-computed data from host to device
    hipMemcpy(deviceCubedDigits, CUBED_DIGITS, CUBED_DIGITS_BYTES, hipMemcpyHostToDevice);

    // Bind pre-computed data to texture reference on device
    hipBindTexture(0, CUBED_DIGITS_DEVICE_TEXTURE, deviceCubedDigits, CUBED_DIGITS_BYTES);

    // Declare streams
    hipStream_t streams[NUM_STREAMS];

    // Create streams
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    // Declare event variables to measure execution time
    float elapsedTime_1, elapsedTime_2;
    hipEvent_t startTime_1, startTime_2, endTime_1, endTime_2;

    // Create events to measure execution time
    hipEventCreate(&startTime_1);
    hipEventCreate(&startTime_2);
    hipEventCreate(&endTime_1);
    hipEventCreate(&endTime_2);

    // Define kernel configuration variables
    dim3 blockDim(BLOCK_DIM);
    dim3 gridDim((ARRAY_DIM - 1) / blockDim.x + 1);

    // Launch three-digit Armstrong numbers calculation kernel on device and record start of execution
    ThreeDigitArmstrongNumbersKernel<<<gridDim, blockDim, 0, streams[0]>>>(deviceArray, 0);
    hipEventRecord(startTime_1, streams[0]);
    ThreeDigitArmstrongNumbersKernel<<<gridDim, blockDim, 0, streams[1]>>>(deviceArray, 1);
    hipEventRecord(startTime_2, streams[1]);

    // Synchronize start of execution calls
    hipEventSynchronize(startTime_1);
    hipEventSynchronize(startTime_2);

    // Record end of execution
    hipEventRecord(endTime_1, streams[0]);
    hipEventRecord(endTime_2, streams[1]);

    // Synchronize end of execution calls
    hipEventSynchronize(endTime_1);
    hipEventSynchronize(endTime_2);

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime_1, startTime_1, endTime_1);
    hipEventElapsedTime(&elapsedTime_2, startTime_2, endTime_2);
    std::cout << "Elapsed Time on Device Stream №1: " << elapsedTime_1 << " ms\n";
    std::cout << "Elapsed Time on Device Stream №2: " << elapsedTime_2 << " ms\n";

    // Destroy events
    hipEventDestroy(startTime_1);
    hipEventDestroy(startTime_2);
    hipEventDestroy(endTime_1);
    hipEventDestroy(endTime_2);

    // Transfer output data from device to host
    hipMemcpy(hostArray, deviceArray, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // Print output data on host
    std::cout << "Three-digit Armstrong numbers computed on device: ";
    for (unsigned number = 0; number < ARRAY_DIM; ++number) {
        if (hostArray[number]) {
            std::cout << number << ' ';
        }
    }
    std::cout << '\n';

    // Unbind pre-computed data to texture reference on device
    hipUnbindTexture(CUBED_DIGITS_DEVICE_TEXTURE);

    // Free device memory
    hipFree(deviceArray);

    // Check for errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }

    return exitStatus;
}