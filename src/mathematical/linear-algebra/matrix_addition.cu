
#include <hip/hip_runtime.h>
#include <iostream>

// Define global constants
constexpr unsigned NUM_ROWS = 1 << 8;
constexpr unsigned NUM_COLS = 1 << 9;
constexpr unsigned BLOCK_DIM = 1 << 5;
constexpr unsigned NUM_STREAMS = 1 << 1;
constexpr unsigned TOTAL_SIZE = NUM_ROWS * NUM_COLS;
constexpr unsigned STREAM_SIZE = TOTAL_SIZE / NUM_STREAMS;
constexpr unsigned TOTAL_BYTES = TOTAL_SIZE * sizeof(float);
constexpr unsigned STREAM_BYTES = TOTAL_BYTES / NUM_STREAMS;

// Define matrix addition kernel
__global__ void MatrixAdditionKernel(float *A, float *B, float *C, unsigned streamIdx) {
    unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx = row * NUM_COLS + col;
    if (row < NUM_ROWS && col < NUM_COLS &&
        (streamIdx == 0 && idx <= STREAM_SIZE ||
        streamIdx == 1 && idx > STREAM_SIZE)) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    // Declare pointers to input and output data on host
    float *hostA = nullptr, *hostB = nullptr, *hostC = nullptr;

    // Allocate pinned host memory for input data
    hipHostMalloc(&hostA, TOTAL_BYTES, hipHostMallocDefault);
    hipHostMalloc(&hostB, TOTAL_BYTES, hipHostMallocDefault);
    hipHostMalloc(&hostC, TOTAL_BYTES, hipHostMallocDefault);

    // Initialize input data on host
    for (unsigned row = 0; row < NUM_ROWS; ++row) {
        for (unsigned col = 0; col < NUM_COLS; ++col) {
            hostA[row * NUM_COLS + col] = 2.0f;
        }
    }
    for (unsigned row = 0; row < NUM_ROWS; ++row) {
        for (unsigned col = 0; col < NUM_COLS; ++col) {
            hostB[row * NUM_COLS + col] = 3.0f;
        }
    }
    
    // Declare pointers to input and output data on device
    float *deviceA = nullptr, *deviceB = nullptr, *deviceC = nullptr;

    // Allocate device memory for input and output data
    hipMalloc(&deviceA, TOTAL_BYTES);
    hipMalloc(&deviceB, TOTAL_BYTES);
    hipMalloc(&deviceC, TOTAL_BYTES);

    // Declare streams
    hipStream_t streams[NUM_STREAMS];

    // Create streams
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    // Copy input data from host to device
    hipMemcpyAsync(deviceA, hostA, STREAM_BYTES, hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(deviceB, hostB, STREAM_BYTES, hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(deviceA + STREAM_SIZE, hostA + STREAM_SIZE, STREAM_BYTES, hipMemcpyHostToDevice, streams[1]);
    hipMemcpyAsync(deviceB + STREAM_SIZE, hostB + STREAM_SIZE, STREAM_BYTES, hipMemcpyHostToDevice, streams[1]);

    // Declare event variables to measure execution time
    float elapsedTime_1, elapsedTime_2;
    hipEvent_t startTime_1, startTime_2, endTime_1, endTime_2;

    // Create events to measure execution time
    hipEventCreate(&startTime_1);
    hipEventCreate(&startTime_2);
    hipEventCreate(&endTime_1);
    hipEventCreate(&endTime_2);

    // Define kernel configuration variables
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((NUM_ROWS - 1) / blockDim.x + 1, (NUM_COLS - 1) / blockDim.y + 1);

    // Launch matrix addition kernels on device and record start of execution
    MatrixAdditionKernel<<<gridDim, blockDim, 0, streams[0]>>>(deviceA, deviceB, deviceC, 0);
    hipEventRecord(startTime_1, streams[0]);
    MatrixAdditionKernel<<<gridDim, blockDim, 0, streams[1]>>>(deviceA, deviceB, deviceC, 1);
    hipEventRecord(startTime_2, streams[1]);
    
    // Synchronize start of execution calls
    hipEventSynchronize(startTime_1);
    hipEventSynchronize(startTime_2);

    // Record end of execution
    hipEventRecord(endTime_1, streams[0]);
    hipEventRecord(endTime_2, streams[1]);

    // Synchronize end of execution calls
    hipEventSynchronize(endTime_1);
    hipEventSynchronize(endTime_2);

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime_1, startTime_1, endTime_1);
    hipEventElapsedTime(&elapsedTime_2, startTime_2, endTime_2);
    std::cout << "Elapsed Time on Device Stream №1: " << elapsedTime_1 << " ms\n";
    std::cout << "Elapsed Time on Device Stream №2: " << elapsedTime_2 << " ms\n";

    // Destroy events
    hipEventDestroy(startTime_1);
    hipEventDestroy(startTime_2);
    hipEventDestroy(endTime_1);
    hipEventDestroy(endTime_2);

    // Transfer output data from device to host
    hipMemcpyAsync(hostC, deviceC, STREAM_BYTES, hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(hostC + STREAM_SIZE, deviceC + STREAM_SIZE, STREAM_BYTES, hipMemcpyDeviceToHost, streams[1]);

    // Destroy streams
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);

    // Print output data on host
    std::cout << "C = A + B:\n";
    for (unsigned row = 0; row < NUM_ROWS; ++row) {
        for (unsigned col = 0; col < NUM_COLS; ++col) {
            std::cout << hostC[row * NUM_COLS + col] << ' ';
        }
        std::cout << '\n';
    }

    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    // Free pinned host memory
    hipHostFree(hostA);
    hipHostFree(hostB);
    hipHostFree(hostC);

    // Check for errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }

    return exitStatus;
}