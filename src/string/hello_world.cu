
#include <hip/hip_runtime.h>
#include <iostream>

// Define global constants
constexpr unsigned BLOCK_DIM = 1 << 5;
constexpr unsigned NUM_STREAMS = 1 << 1;
constexpr char *GREETING = "Hello World";
constexpr unsigned NUM_GREETINGS = BLOCK_DIM;
constexpr unsigned NUM_GREETINGS_PER_STREAM = NUM_GREETINGS / NUM_STREAMS;

// Define hello world kernel
__global__ void HelloWorldKernel(unsigned streamIdx) {
    unsigned idx = NUM_GREETINGS_PER_STREAM * streamIdx + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NUM_GREETINGS) {
        printf("%s №%d!\n", GREETING, idx);
    }
}

int main() {
    // Declare streams
    hipStream_t streams[NUM_STREAMS];

    // Create streams
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    // Declare event variables to measure execution time
    float elapsedTime_1, elapsedTime_2;
    hipEvent_t startTime_1, startTime_2, endTime_1, endTime_2;

    // Create events to measure execution time
    hipEventCreate(&startTime_1);
    hipEventCreate(&startTime_2);
    hipEventCreate(&endTime_1);
    hipEventCreate(&endTime_2);

    // Define kernel configuration variables
    dim3 gridDim(1);
    dim3 blockDim(BLOCK_DIM);
    
    // Launch hello world kernel on device and record start of execution
    HelloWorldKernel<<<gridDim, blockDim, 0, streams[0]>>>(0);
    hipEventRecord(startTime_1, streams[0]);
    HelloWorldKernel<<<gridDim, blockDim, 0, streams[1]>>>(1);
    hipEventRecord(startTime_2, streams[1]);

    // Synchronize start of execution calls
    hipEventSynchronize(startTime_1);
    hipEventSynchronize(startTime_2);

    // Record end of execution
    hipEventRecord(endTime_1, streams[0]);
    hipEventRecord(endTime_2, streams[1]);

    // Synchronize end of execution calls
    hipEventSynchronize(endTime_1);
    hipEventSynchronize(endTime_2);

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime_1, startTime_1, endTime_1);
    hipEventElapsedTime(&elapsedTime_2, startTime_2, endTime_2);
    std::cout << "Elapsed Time on Device Stream №1: " << elapsedTime_1 << " ms\n";
    std::cout << "Elapsed Time on Device Stream №2: " << elapsedTime_2 << " ms\n";

    // Destroy events
    hipEventDestroy(startTime_1);
    hipEventDestroy(startTime_2);
    hipEventDestroy(endTime_1);
    hipEventDestroy(endTime_2);

    // Check for any errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }
    
    return exitStatus;
}