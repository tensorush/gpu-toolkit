
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>

// Define global constants
constexpr float PI = 3.1415927f;
constexpr unsigned NUM_SPHERES = 8;
constexpr unsigned BLOCK_DIM = 1 << 5;
constexpr unsigned NUM_SAMPLES = 1 << 15;
constexpr unsigned IMAGE_WIDTH = 1 << 10;
constexpr unsigned IMAGE_HEIGHT = 1 << 10;
constexpr unsigned NUM_RAY_BOUNCES = 1 << 2;
constexpr unsigned NUM_PIXELS = IMAGE_WIDTH * IMAGE_HEIGHT;
constexpr unsigned IMAGE_BYTES = NUM_PIXELS * sizeof(float3);

// Define vector operations on float3
__device__ float3 Scale(const float3 &vector, const float scalar) {
	return make_float3(scalar * vector.x, scalar * vector.y, scalar * vector.z);
}

__device__ float3 Add(const float3 &vector1, const float3 &vector2) {
	return make_float3(vector1.x + vector2.x, vector1.y + vector2.y, vector1.z + vector2.z);
}

__device__ float3 Subtract(const float3 &vector1, const float3 &vector2) {
	return make_float3(vector1.x - vector2.x, vector1.y - vector2.y, vector1.z - vector2.z);
}

__device__ float3 Multiply(const float3 &vector1, const float3 &vector2) {
	return make_float3(vector1.x * vector2.x, vector1.y * vector2.y, vector1.z * vector2.z);
}

__device__ float DotProduct(const float3 &vector1, const float3 &vector2) {
	return vector1.x * vector2.x + vector1.y * vector2.y + vector1.z * vector2.z;
}

__device__ float3 Normalize(const float3 &vector) {
	return Scale(vector, 1.0f / std::sqrt(DotProduct(vector, vector)));
}

__device__ float3 CrossProduct(const float3 &vector1, const float3 &vector2) {
	return make_float3(vector1.y * vector2.z - vector1.z * vector2.y, vector1.z * vector2.x - vector1.x * vector2.z, vector1.x * vector2.y - vector1.y * vector2.x);
}

// Define helper functions
__host__ __device__ float clampBetweenZeroAndOne(const float x) {
    return (x < 0.0f) ? (0.0f) : ((x > 1.0f) ? (1.0f) : (x));
}

__host__ unsigned convertColourFromFloatToInt(const float x) {
    return static_cast<unsigned>(std::pow(clampBetweenZeroAndOne(x), 1.0f / 2.2f) * 255.0f + 0.5f);
}

// Define rendering structures
enum MaterialType {
    DIFFUSE
};

struct Ray {
    float3 origin;
    float3 direction;
    __device__ Ray(float3 origin_, float3 direction_) : origin(origin_), direction(direction_) {}
};

struct Sphere {
    float radius;
    float3 centre;
    float3 colour;
    float3 emission;
    MaterialType materialType;
    __device__ float computeHitDistanceFromRayOriginToSphere(const Ray &ray) const {
        /*
            Ray equation:
                hitPoint = rayOrigin + hitDistance * rayDirection
            Sphere equation:
                (hitPoint - sphereCentre) · (hitPoint - sphereCentre) - sphereRadius^2 = 0
            Substitute hitPoint and solve resulting quadratic equation for shortest hitDistance:
                (rayDirection · rayDirection) * hitDistance^2
                + 2 * rayDirection · (rayOrigin - sphereCentre) * hitDistance
                + (rayOrigin - sphereCentre) · (rayOrigin - sphereCentre) - sphereRadius^2 = 0
        */
        // Declare ray-to-sphere hit distance
        float hitDistance;
        // Define epsilon to aid floating point imprecision
        float epsilon = 1e-4f;
        // Distance from ray origin to sphere centre
        float3 raySphereDistance = Subtract(centre, ray.origin);
        // Find b coefficient of quadratic equation
        float b = DotProduct(raySphereDistance, ray.direction);
        // Find discriminant of quadratic equation
        float discriminant = b * b - DotProduct(raySphereDistance, raySphereDistance) + radius * radius;
        // If discriminant < 0, no intersections exist
        if (discriminant < 0.0f) {
            return 0.0f;
        }
        // If discriminant >= 0, find shortest hit distance
        discriminant = std::sqrt(discriminant);
        return ((hitDistance = b - discriminant) > epsilon) ? (hitDistance) : (((hitDistance = b + discriminant) > epsilon) ? (hitDistance) : (0.0f));
    }
};

// Define Cornell Box made of spheres in device memory
__constant__ Sphere SPHERES[] = {
    { 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.75f, 0.75f, 0.75f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Floor
    { 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.75f, 0.75f, 0.75f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Back Wall
    { 16.5f, { 73.0f, 16.5f, 78.0f }, { 1.0f, 1.0f, 1.0f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Small Left
    { 16.5f, { 27.0f, 16.5f, 47.0f }, { 1.0f, 1.0f, 1.0f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Small Right
    { 600.0f, { 50.0f, 680.83f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFFUSE }, // Light Source
    { 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.75f, 0.25f, 0.25f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Left Wall
    { 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.75f, 0.75f, 0.75f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Ceiling
    { 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.25f, 0.25f, 0.75f }, { 0.0f, 0.0f, 0.0f }, DIFFUSE }, // Right Wall
};

// Define rendering kernels
__device__ bool DoesRayIntersectSphere(const Ray &ray, float &hitDistance, unsigned &hitSphereIndex){
    hitDistance = INFINITY;
    float smallerHitDistance;
    // Check every sphere for ray hit
    for (unsigned sphereIndex = 0; sphereIndex < NUM_SPHERES; ++sphereIndex) {
        // Update hit distance and hit sphere index
        if ((smallerHitDistance = SPHERES[sphereIndex].computeHitDistanceFromRayOriginToSphere(ray)) && smallerHitDistance < hitDistance) {
            hitDistance = smallerHitDistance;
            hitSphereIndex = sphereIndex;
        }
    }
    return hitDistance < INFINITY;
}

__device__ float GenerateRandomNumber(unsigned *seed1, unsigned *seed2) {
    *seed1 = 36'969 * ((*seed1) & 65'535) + ((*seed1) >> 16);
    *seed2 = 18'000 * ((*seed2) & 65'535) + ((*seed2) >> 16);
    union {
        float floatType;
        unsigned unsignedType;
    } hash;
    hash.unsignedType = ((((*seed1) << 16) + (*seed2)) & 8'388'607) | 1'073'741'824;
    return (hash.floatType - 2.0f) / 2.0f;
}

__device__ float3 TraceRayPath(Ray &ray, unsigned *seed1, unsigned *seed2) {
    /*
        Rendering equation:
            outgoingRadiance = emittedRadiance + reflectedRadiance,
            where reflectedRadiance is an integral of incoming radiance over the hemisphere above the hit point
            multiplied by the BRDF of the hit material and the geometry term - cosine of incident angle
    */
    // Define accumulated ray colour to be black
    float3 accumulatedRayColour = make_float3(0.0f, 0.0f, 0.0f);
    // Define colour bleeding factor to be neutral
    float3 colourBleedingFactor = make_float3(1.0f, 1.0f, 1.0f);
    // Bounce ray around scene
    for (unsigned bounceIdx = 0; bounceIdx < NUM_RAY_BOUNCES; ++bounceIdx) {
        float hitDistance;
        unsigned hitSphereIndex;
        // Check every sphere for ray hit
        if (!DoesRayIntersectSphere(ray, hitDistance, hitSphereIndex)) {
            return make_float3(0.0f, 0.0f, 0.0f);
        }
        // Compute hit point and normal
        const Sphere &hitSphere = SPHERES[hitSphereIndex];
        float3 hitPoint = Add(ray.origin, Scale(ray.direction, hitDistance));
        float3 hitNormal = Normalize(Subtract(hitPoint, hitSphere.centre));
        if (DotProduct(hitNormal, ray.direction) > 0.0f) {
            hitNormal = Scale(hitNormal, -1.0f);
        }
        // Add emitted light to accumulated ray colour
        accumulatedRayColour = Add(accumulatedRayColour, Multiply(colourBleedingFactor, hitSphere.emission));
        // Generate random azimuth and zenith angles for new ray direction
        float azimuth = 2.0f * PI * GenerateRandomNumber(seed1, seed2);
        float zenith = GenerateRandomNumber(seed1, seed2);
        float squareRoottOfZenith = std::sqrt(zenith);
        // Construct orthonormal basis to generate random ray direction
        float3 unitNormal1 = hitNormal;
        float3 someNormal = (std::abs(unitNormal1.x) > 0.1f) ? (make_float3(0.0f, 1.0f, 0.0f)) : (make_float3(1.0f, 0.0f, 0.0f));
        float3 unitNormal2 = Normalize(CrossProduct(someNormal, unitNormal1));
        float3 unitNormal3 = CrossProduct(unitNormal1, unitNormal2);
        // Generate random ray direction on hemisphere using polar coordinates
        // and cosine weighted importance sampling, which favours ray directions closer to normal
        unitNormal1 = Scale(unitNormal1, std::sqrt(1.0f - zenith));
        unitNormal2 = Scale(unitNormal2, std::cos(azimuth) * squareRoottOfZenith);
        unitNormal3 = Scale(unitNormal3, std::sin(azimuth) * squareRoottOfZenith);
        ray.direction = Normalize(Add(unitNormal1, Add(unitNormal2, unitNormal3)));
        // Offset ray origin slightly to prevent self intersection
        ray.origin = Add(hitPoint, Scale(hitNormal, 0.05f));
        // Weight by sphere colour
        colourBleedingFactor = Multiply(colourBleedingFactor, hitSphere.colour);
        // Weight light contribution by cosine of angle between outgoing light and normal
        colourBleedingFactor = Scale(colourBleedingFactor, DotProduct(ray.direction, hitNormal));
        // Weight by BRDF fudge factor
        colourBleedingFactor = Scale(colourBleedingFactor, 2.0f);
    }
    return accumulatedRayColour;
}

__global__ void PathTracingKernel(float3 *image) {
    // Assign each thread to pixel
    unsigned pixelCoordinateX = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned pixelCoordinateY = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned curPixel = (IMAGE_HEIGHT - pixelCoordinateY - 1) * IMAGE_WIDTH + pixelCoordinateX;
    unsigned seed1 = pixelCoordinateX;
    unsigned seed2 = pixelCoordinateY;
    // Generate ray directed at lower left image corner
    Ray ray(make_float3(50.0f, 52.0f, 295.6f), Normalize(make_float3(0.0f, -0.042612f, -1.0f)));
    // Compute directions for other rays by adding offsets along x and y pixel coordinate axes
    float fieldOfViewAngle = 0.5135f;
    float3 pixelColour = make_float3(0.0f, 0.0f, 0.0f);
    float3 rayOffsetX = make_float3(IMAGE_WIDTH * fieldOfViewAngle / IMAGE_HEIGHT, 0.0f, 0.0f);
    float3 rayOffsetY = Scale(Normalize(CrossProduct(rayOffsetX, ray.direction)), fieldOfViewAngle);
    // Sample rays
    for (unsigned sampleIdx = 0; sampleIdx < NUM_SAMPLES; ++sampleIdx) {
        // Compute primary ray direction
        float3 direction = Add(ray.direction, Add(Scale(rayOffsetX, (pixelCoordinateX + 0.25f) / IMAGE_WIDTH - 0.5f), Scale(rayOffsetY, (pixelCoordinateY + 0.25f) / IMAGE_HEIGHT - 0.5f)));
        // Create primary ray
        Ray primaryRay = Ray(Add(ray.origin, Scale(direction, 40.0f)), Normalize(direction));
        // Add traced ray path to pixel color
        pixelColour = Add(pixelColour, Scale(TraceRayPath(primaryRay, &seed1, &seed2), 1.0f / NUM_SAMPLES));
    }
    // Clamp floating-point pixel colour in range [0; 1]
    image[curPixel] = make_float3(clampBetweenZeroAndOne(pixelColour.x), clampBetweenZeroAndOne(pixelColour.y), clampBetweenZeroAndOne(pixelColour.z));
}

int main() {
    // Declare pointer for output image on host
    float3 *hostImage = nullptr;

    // Allocate host memory for output image
    hipHostMalloc(&hostImage, IMAGE_BYTES, hipHostMallocDefault);

    // Declare pointer for output image on device
    float3 *deviceImage = nullptr;

    // Allocate device memory for output image
    hipMalloc(&deviceImage, IMAGE_BYTES);

    // Declare event variables to measure execution time
    float elapsedTime;
    hipEvent_t startTime, endTime;

    // Create events to measure execution time
    hipEventCreate(&startTime);
    hipEventCreate(&endTime);

    // Define kernel configuration variables
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((IMAGE_WIDTH - 1) / blockDim.x + 1, (IMAGE_HEIGHT - 1) / blockDim.y + 1);

    // Record start of execution
    hipEventRecord(startTime, 0);
    
    // Synchronize start of execution call
    hipEventSynchronize(startTime);

    // Launch path tracing kernel on device
    PathTracingKernel<<<gridDim, blockDim>>>(deviceImage);

    // Record end of execution
    hipEventRecord(endTime, 0);

    // Synchronize end of execution call
    hipEventSynchronize(endTime);

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime, startTime, endTime);
    std::cout << "Elapsed Time on Device at " << IMAGE_HEIGHT << 'x' << IMAGE_WIDTH << " resolution: " << elapsedTime << " ms\n";

    // Destroy events
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);

    // Copy output image from device to host
    hipMemcpy(hostImage, deviceImage, IMAGE_BYTES, hipMemcpyDeviceToHost);

    // Open output file
    std::ofstream imageFile;
    imageFile.open("path_traced_image.ppm");

    // Write output image to .ppm file
    imageFile << "P3\n" << IMAGE_WIDTH << ' ' << IMAGE_HEIGHT << '\n' << 255 << '\n';
    for (unsigned pixelIdx = 0; pixelIdx < NUM_PIXELS; ++pixelIdx) {
        imageFile << convertColourFromFloatToInt(hostImage[pixelIdx].x) << ' ' << convertColourFromFloatToInt(hostImage[pixelIdx].y) << ' ' << convertColourFromFloatToInt(hostImage[pixelIdx].z) << ' ';
    }

    // Close output file
    imageFile.close();

    // Free device memory
    hipFree(deviceImage);

    // Free host memory
    hipHostFree(hostImage);

    // Check for any errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }
    
    return exitStatus;
}